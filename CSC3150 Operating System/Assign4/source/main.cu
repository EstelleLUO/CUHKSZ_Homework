#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string.h>

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

//Volume Control Block, which constains volume details
//Including #of blocks, #of free blocks, block size, free block pointers or array
#define SUPERBLOCK_SIZE 			4096 //4KB
//File Control Block, which is a Storage Structure consisting of information about a file
#define FCB_SIZE 					32 //32 bytes per FCB
#define FCB_ENTRIES 				1024
//Total size of avaible memory
#define STORAGE_SIZE 				1085440 //1060KB
#define STORAGE_BLOCK_SIZE 			32

#define MAX_FILENAME_SIZE 			20 //20 bytes
//max number of files 
#define MAX_FILE_NUM 				1024
//max size of file
#define MAX_ONE_FILE_SIZE 			1024
//max size of file name
#define MAX_FILE_NAME 				20

//The maximum size of file memory
#define MAX_FILE_SIZE 				1048576 //1024KB
#define BIT_TO_BYTE 				8 //Used in the conversion of bit to byte
//the start position of file memory
#define FILE_STORAGE_START			(STORAGE_SIZE-MAX_FILE_SIZE)
#define BITS_IN_BYTE 				8

#define OP_ERROR 					-1

#define TRUE 						1
#define FALSE 						0

//for valid/invalid FCB and valid/free block
#define VALID 						1
#define FREE 						0
#define INVALID 					0

//for bitmap
#define FREE_BLOCK_MASK 			0x1
#define FREE_BLOCK_BIT 				0
#define NON_FREE_BLOCK_MASK 		1

//used as read/write flag
#define G_READ 						0
#define G_WRITE 					1

//gsys flags
#define RM 							0
#define LS_D 						1
#define LS_S 						2

typedef unsigned char uchar;
typedef uint32_t u32;

//storing syatem information
typedef struct {
	uchar bitmap[SUPERBLOCK_SIZE];	       	//the bitmap recording free blocks
	u32 file_num ;	 					 	//number of files in the file system
	u32 file_list_time[MAX_FILE_NUM];	    //list files in order of decreasing modified time
	u32 file_list_size[MAX_FILE_NUM];	    //list files in order of decreasing file size
}FileSystem;

//FCB entry
typedef struct {
	char name[MAX_FILE_NAME];				//file name
	u32 valid_entry ;						//indicate whether this entry is valid
	u32 op ;								//the allowed operations of the file
	u32 time ;								//the last modified time of a file
	u32 block_num ;							//the index of its file block
	u32 file_size ;							//the size of a file
}FCB;

//FCB array pointer
__device__ FCB *fcb_table;

//system struct pointer
__device__ FileSystem *file_system;

//total storage 
__device__ uchar volume_d[STORAGE_SIZE];

/* Get the values for bitmap*/
__device__ u32 get_bitmap(u32 index)
{
	u32 start_pos = index / BITS_IN_BYTE;
	u32 offset = index%BITS_IN_BYTE;
	return ((file_system->bitmap[start_pos]) >> offset)&FREE_BLOCK_MASK;
}

/* Set the values for bitmap*/
__device__ void set_bitmap(u32 index, u32 flag)
{
	u32 start_pos = index / BITS_IN_BYTE;
	u32 offset = index%BITS_IN_BYTE;
	if (flag == VALID) file_system->bitmap[start_pos] = file_system->bitmap[start_pos] | (VALID << offset);
	else file_system->bitmap[start_pos] = file_system->bitmap[start_pos] & (~(VALID << offset));
}

/* Compare if two file names are the same */
__device__ bool compare_name(const char *dest, const char *src)
{
	int index = 0;
	while (index<MAX_FILE_NAME) {
		if (src[index] != dest[index]) return false;
		else if (src[index] == '\0' && dest[index] == '\0') return true;
		index++;
	}
	return true;
}


/* Copy file names*/
__device__ void cpy_filename(char *dest, const char *src)
{
	u32 index = 0;
	while (src[index] != '\0') {
		if (index<MAX_FILE_NAME) dest[index] = src[index];
		else {
			printf("The file name exceeds the maximum file name length\n");
			break;
		}
		index++;
	}
	dest[index] = '\0';
}

/* Open Function Implementation*/
__device__ u32 open(const char *s, int op)
{
	/* Implement open operation here */
	//the index of the FCB entry of a file
	u32 file_fcb = -1;
	//the index of a free FCB entry
	u32 free_fcb = -1;
	//Find file is whether exist in FCB or not
	for (int i = 0; i<MAX_FILE_NUM; i++) {
		if (compare_name(fcb_table[i].name, s) ) {
			//If found
			if (fcb_table[i].valid_entry == VALID) {
				file_fcb = i;
				fcb_table[i].op = op;
				return file_fcb;
			}
		}
		if (fcb_table[i].valid_entry == FREE) free_fcb = i;
	}

	//if not found
	if (file_fcb ==-1) {
		//the index of a free block
		u32 free_block = -1;
		//search the bitmap for free block
		for (int i = 0; i<MAX_FILE_NUM; i++) {
			if (get_bitmap(i) == FREE) {
				free_block = i;
				break;
			}
		}
		//if there is a free block, create a new FCB and record its block number
		if (free_block != -1) {
			//renew modified time of other files
			for (int i = 0; i<MAX_FILE_NUM; i++) {
				if (fcb_table[i].valid_entry == VALID) fcb_table[i].time++;
			}
			//set the modified time of the new file
			fcb_table[free_fcb].time = 0;
			fcb_table[free_fcb].op = op;
			//set the FCB entry valid
			fcb_table[free_fcb].valid_entry = TRUE;
			//make FCB point to the free block
			fcb_table[free_fcb].block_num = free_block;
			//set the file name
			cpy_filename(fcb_table[free_fcb].name, s);
			//set bitmap to indicate it is occupied
			set_bitmap(free_block, VALID);
			//renew total number of files
			file_system->file_num++;

			//renew file lists
			file_system->file_list_time[file_system->file_num - 1] = free_fcb;
			file_system->file_list_size[file_system->file_num - 1] = free_fcb;

			return free_fcb;
		}
		//if no free blocks are available, return error;
		else {
			printf("no free block\n");
			return OP_ERROR;
		}
	}
}

/* Remove Function Implementation */
__device__ void rm(const char *fileName)
{
	//the FCB entry of the to-be-removed file
	u32 file_fcb = -1;
	//search for FCB by file name
	for (int i = 0; i<MAX_FILE_NUM; i++) {
		if (compare_name(fcb_table[i].name, fileName)) {
			file_fcb = i;
			break;
		}
	}
	//if found
	if (file_fcb != -1) {
		//the real position of the file
		u32 file_start = fcb_table[file_fcb].block_num*FCB_ENTRIES;
		//the modified time of the file
		u32 time = fcb_table[file_fcb].time;
		u32 flag = FALSE;

		//remove file in file list
		for (int i = 0; i<file_system->file_num; i++) {
			if (file_system->file_list_time[i] == file_fcb) flag = TRUE;
			if (flag == TRUE && i != file_system->file_num - 1)
				file_system->file_list_time[i] = file_system->file_list_time[i + 1];
			else if (flag == TRUE && i == file_system->file_num - 1)
				file_system->file_list_time[i] = 0;
		}
		flag = FALSE;
		for (int i = 0; i<file_system->file_num; i++) {
			if (file_system->file_list_size[i] == file_fcb) flag = TRUE;
			if (flag == TRUE && i != file_system->file_num - 1)
				file_system->file_list_size[i] = file_system->file_list_size[i + 1];
			else if (flag == TRUE && i == file_system->file_num - 1)
				file_system->file_list_size[i] = 0;
		}

		//reset system info
		file_system->file_num--;
		set_bitmap(fcb_table[file_fcb].block_num, FREE);
		//reset modified time of other files
		for (int i = 0; i<MAX_FILE_NUM; i++) {
			if (fcb_table[i].valid_entry == VALID) {
				if (fcb_table[i].time>time)
					fcb_table[i].time--;
			}
		}
		//clear file content
		for (int i = 0; i<MAX_ONE_FILE_SIZE; i++)
			volume_d[FILE_STORAGE_START + file_start + i] = 0;
		//reset FCB block
		fcb_table[file_fcb].valid_entry = FREE;
		fcb_table[file_fcb].op = G_READ;
		fcb_table[file_fcb].time = 0;
		fcb_table[file_fcb].block_num = 0;
		fcb_table[file_fcb].file_size = 0;
		for (int j = 0; j<MAX_FILE_NAME; j++)
			fcb_table[file_fcb].name[j] = 0;

	}
	//if not found
	else printf("Cannot find file %s\n", fileName);
}

/* Write Function Implementation */
__device__ u32 write(const uchar *input, u32 size, u32 fp)
{
	//if file is not in write op, return error
	if (fcb_table[fp].op != G_WRITE) {
		printf("%s is not in write op\n", fcb_table[fp].name);
		return OP_ERROR;
	}
	u32 file_start = fcb_table[fp].block_num*FCB_ENTRIES;
	u32 previous_time = fcb_table[fp].time;
	u32 count;
	if (size < MAX_ONE_FILE_SIZE) count = size;
	else count = MAX_ONE_FILE_SIZE;
	//if bytes to write is more than max file size
	if (size>MAX_ONE_FILE_SIZE)
		printf("Cannot write more than 1024 bytes in a file\n");
	//write the file
	for (int i = 0; i<count; i++)
		volume_d[FILE_STORAGE_START + file_start + i] = input[i];
	//renew the file size in FCB
	fcb_table[fp].file_size = count;
	//renew modified time in FCB
	for (int i = 0; i<MAX_FILE_NUM; i++) {
		if (fcb_table[i].valid_entry == VALID && fcb_table[i].time <= previous_time) {
			fcb_table[i].time++;
		}
	}
	fcb_table[fp].time = 0;

	//renew file lists
	for (int i = 0; i<file_system->file_num - 1; i++) {
		if (fcb_table[file_system->file_list_time[i]].time <= previous_time)
			file_system->file_list_time[i] = file_system->file_list_time[i + 1];
	}
	file_system->file_list_time[file_system->file_num - 1] = fp;

	u32 start_idx = -1;
	u32 end_idx = file_system->file_num;
	u32 flag = FALSE;
	for (int i = 0; i<file_system->file_num; i++) {
		if (file_system->file_list_size[i] == fp) start_idx = i;
		if (fcb_table[file_system->file_list_size[i]].file_size <= count && flag == FALSE){
			flag = TRUE;
			end_idx = i;
		}
	}
	//if we don't find the final position, set it the tail of list
	if (end_idx>start_idx) {
		u32 temp = file_system->file_list_size[start_idx];
		for (int i = start_idx; i<end_idx - 1; i++)
			file_system->file_list_size[i] = file_system->file_list_size[i + 1];
		file_system->file_list_size[end_idx - 1] = temp;
	}
	else if (end_idx<start_idx) {
		u32 temp = file_system->file_list_size[start_idx];
		for (int i = start_idx; i>end_idx; i--)
			file_system->file_list_size[i] = file_system->file_list_size[i - 1];
		file_system->file_list_size[end_idx] = temp;
	}

	//return number of bytes written
	return count;
}

/* Read Function Implementation */
__device__ u32 read(uchar *output, u32 size, u32 fp)
{
	//if file is not in read op, return error
	if (fcb_table[fp].op != G_READ) {
		printf("%s is not in read op\n", fcb_table[fp].name);
		return OP_ERROR;
	}

	u32 file_start = fcb_table[fp].block_num*FCB_ENTRIES;
	u32 count;
	if (size < fcb_table[fp].file_size) count = size;
	else count = fcb_table[fp].file_size;
	//if bytes to read is more than max file size
	if (size>fcb_table[fp].file_size)
		printf("Cannot read more than file size\n");
	//read the file
	for (int i = 0; i<count; i++)
		output[i] = volume_d[FILE_STORAGE_START + file_start + i];
	//return number of bytes read
	return count;
}


/* LS_D and LS_S Implementation */
__device__ void gsys(int op)
{
char *name;
u32 size=0;
/* Implement LS_D and LS_S operation here */
//LS_D Operation
if (op==LS_D){
printf("===sort by modified time===\n");

for(int i=file_system->file_num-1;i>=0;i--){
name = fcb_table[file_system->file_list_time[i]].name;
printf("%s\n",name);
}
}
//LS_S Operation
else if(op==LS_S){
printf("===sort by file size===\n");

for(int i=0;i<file_system->file_num;i++){
name=fcb_table[file_system->file_list_size[i]].name;
size=fcb_table[file_system->file_list_size[i]].file_size;
printf("%s %d\n",name,size);
}

}
else printf("The command is invalid\n");
}

/* RM Implementation */
__device__ void gsys(int op, char *s)
{
/* Implement rm operation here */
if (op==RM){
rm(s);
}
else printf("The command is invalid\n");
}


__host__ void write_binaryFile(char *fileName, void *buffer, int bufferSize)
{

FILE *fp;
fp = fopen(fileName, "wb");
fwrite(buffer, 1, bufferSize, fp);
fclose(fp);
}

__host__ int load_binaryFile(char *fileName, void *buffer, int bufferSize)
{
FILE *fp;
fp = fopen(fileName, "rb");

if (!fp)
{
printf("***Unable to open file %s***\n", fileName);
exit(1);
}

//Get file length
fseek(fp, 0, SEEK_END);
int fileLen = ftell(fp);
fseek(fp, 0, SEEK_SET);

if (fileLen > bufferSize)
{
printf("****invalid testcase!!****\n");
printf("****software warrning: the file: %s size****\n", fileName);
printf("****is greater than buffer size****\n");
exit(1);
}

//Read file contents into buffer
fread(buffer, fileLen, 1, fp);
fclose(fp);
return fileLen;
}


__device__ void init_volume()
{
	file_system = (FileSystem *)volume_d;
	fcb_table = (FCB *)(volume_d + sizeof(*file_system));

	for (int i = 0; i<MAX_FILE_NUM / BITS_IN_BYTE; i++)
		file_system->bitmap[i] = 0;
	file_system->file_num = 0;
	for (int i = 0; i<MAX_FILE_NUM; i++) {
		file_system->file_list_time[i] = 0;
		file_system->file_list_size[i] = 0;
	}

	for (int i = 0; i<MAX_FILE_NUM; i++) {
		fcb_table[i].valid_entry = FREE;
		fcb_table[i].op = G_READ;
		fcb_table[i].time = 0;
		fcb_table[i].block_num = 0;
		fcb_table[i].file_size = 0;
		for (int j = 0; j<MAX_FILE_NAME; j++) {
			fcb_table[i].name[j] = 0;
		}
	}
}

__global__ void mykernel(uchar *input, uchar *output)
{
	init_volume();
	/**************************************
	* Test Case 1
	***************************************/
	// kernel test start  
	u32 fp = open("t.txt\0", G_WRITE);
	write(input, 64, fp);

	fp = open("b.txt\0", G_WRITE);
	write(input + 32, 32, fp);

	fp = open("t.txt\0", G_WRITE);
	write(input + 32, 32, fp);

	fp = open("t.txt\0", G_READ);
	read(output, 32, fp);

	gsys(LS_D);
	gsys(LS_S);

	fp = open("b.txt\0", G_WRITE);
	write(input + 64, 12, fp);

	gsys(LS_S);
	gsys(LS_D);
	gsys(RM, "t.txt\0");
	gsys(LS_S);
	// kernel test end
	/*/
	/**************************************
	* Test Case 2
	**************************************
	//kernel test start

	u32 fp = open("t.txt\0", G_WRITE);
	write(input, 64, fp);

	fp = open("b.txt\0", G_WRITE);
	write(input+32, 32, fp);

	fp = open("t.txt\0", G_WRITE);
	write(input+32, 32, fp);

	fp = open("t.txt\0", G_READ);
	read(output, 32, fp);

	gsys(LS_D);
	gsys(LS_S);

	fp = open("b.txt\0", G_WRITE);
	write(input+64, 12, fp);

	gsys(LS_S);
	gsys(LS_D);
	gsys(RM, "t.txt\0");
	gsys(LS_S);

	char fname[10][20];
	for(int i = 0; i < 10; i++)
	{
	fname[i][0] = i+33;
	for(int j = 1; j < 19; j++)
	fname[i][j] = 64+j;
	fname[i][19] = '\0';
	}
	for(int i = 0; i < 10; i++)
	{
	fp = open(fname[i], G_WRITE);
	write(input+i, 24+i, fp);
	}
	gsys(LS_S);
	for(int i = 0; i < 5; i++)
	gsys(RM, fname[i]);
	gsys(LS_D);
	// kernel test end
	*/
	/**************************************
	* Test Case 3
	**************************************
	//kernel test start
	u32 fp = open("t.txt\0", G_WRITE);
	write(input, 64, fp);
	fp = open("b.txt\0", G_WRITE);
	write(input+32, 32, fp);
	fp = open("t.txt\0", G_WRITE);
	write(input+32, 32, fp);
	fp = open("t.txt\0", G_READ);
	read(output, 32, fp);
	gsys(LS_D);
	gsys(LS_S);
	fp = open("b.txt\0", G_WRITE);
	write(input+64, 12, fp);
	gsys(LS_S);
	gsys(LS_D);
	gsys(RM, "t.txt\0");
	gsys(LS_S);

	char fname[10][20];
	for(int i = 0; i < 10; i++)
	{
	fname[i][0] = i+33;
	for(int j = 1; j < 19; j++)
	fname[i][j] = 64+j;
	fname[i][19] = '\0';
	}
	for(int i = 0; i < 10; i++)
	{
	fp = open(fname[i], G_WRITE);
	write(input+i, 24+i, fp);
	}
	gsys(LS_S);
	for(int i = 0; i < 5; i++)
	gsys(RM, fname[i]);
	gsys(LS_D);
	char fname2[1018][20];
	int p = 0;
	for(int k = 2; k < 15; k++)
	for(int i = 50; i <= 126; i++, p++)
	{
	fname2[p][0] = i;
	for(int j = 1; j < k; j++)
	fname2[p][j] = 64+j;
	fname2[p][k] = '\0';

	}
	for(int i = 0 ; i < 1001; i++)
	{
	fp = open(fname2[i], G_WRITE);
	write(input+i, 24+i, fp);
	}
	gsys(LS_S);

	fp = open(fname2[1000], G_READ);
	read(output+1000, 1024, fp);

	char fname3[17][3];
	for(int i = 0; i < 17; i++)

	{
	fname3[i][0] = 97+i;
	fname3[i][1] = 97+i;
	fname3[i][2] = '\0';
	fp = open(fname3[i], G_WRITE);
	write(input+1024*i, 1024, fp);

	}
	fp = open("EA\0", G_WRITE);
	write(input+1024*100, 1024, fp);
	gsys(LS_S);
	//kernel test end
	*/
}

/************************************************************************************
*
* Main function
*
************************************************************************************/
int main()
{
	uchar *input_h;
	uchar *input;

	uchar *output_h;
	uchar *output;

	input_h = (uchar *)malloc(sizeof(uchar)* MAX_FILE_SIZE);
	output_h = (uchar *)malloc(sizeof(uchar)* MAX_FILE_SIZE);


	hipMalloc(&input, sizeof(uchar)* MAX_FILE_SIZE);
	hipMalloc(&output, sizeof(uchar)* MAX_FILE_SIZE);

	// load binary file from data.bin
	load_binaryFile(DATAFILE, input_h, MAX_FILE_SIZE);


	hipMemcpy(input, input_h, sizeof(uchar)* MAX_FILE_SIZE, hipMemcpyHostToDevice);
	hipMemcpy(output, output_h, sizeof(uchar)* MAX_FILE_SIZE, hipMemcpyHostToDevice);

	mykernel << <1, 1 >> >(input, output);

	hipMemcpy(output_h, output, sizeof(uchar)* MAX_FILE_SIZE, hipMemcpyDeviceToHost);

	// dump output array to snapshot.bin 
	write_binaryFile(OUTFILE, output_h, MAX_FILE_SIZE);

	hipDeviceSynchronize();
	hipDeviceReset();

	return 0;

}
