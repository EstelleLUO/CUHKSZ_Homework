#include <stdio.h>
#include <stdlib.h>
#include <inttypes.h>

#include <hip/hip_runtime.h>


//page size is 32bytes
#define PAGESIZE 32
//32 KB in shared memory 
#define PHYSICAL_MEM_SIZE 32768
//128 KB in global memory
#define STORAGE_SIZE 131072

#define PT_ENTRIES 1024 

#define DATAFILE "./data.bin"
#define OUTFILE "./snapshot.bin"

typedef unsigned char uchar ; 
typedef uint32_t u32 ; 

//page table entries
__device__ int PAGE_ENTRIES = 0;
//count the pagefault times
__device__ int PAGEFAULT_NUM = 0; 

__device__ int currentTime = 0; 

//secondary memory
__device__ uchar storage[STORAGE_SIZE];

//date input and output 
__device__ uchar results[STORAGE_SIZE]; 
__device__ uchar input[STORAGE_SIZE]; 

//Function Declaration
__device__ u32 paging(uchar *buffer, u32 frameNum, u32 offset);
__device__ u32 getPageNum(u32 pt_entries);

//page table
extern __shared__ u32 pt[];
const uint32_t INVALID =0;
const uint32_t VALID =1;
const uint32_t PAGENUMMASK=(1 << 13)-2;

__device__ void init_pageTable( int entries ){
	for(int i = 0; i < entries; i++){
		pt[i] = INVALID; 
	}
}

__device__ uchar Gread( uchar *buffer, u32 addr){
	/* Complete Gread function to read value from data buffer */
	u32 frameNum = addr/PAGESIZE ; 
	u32 offset = addr%PAGESIZE ; 
	
	addr = paging(buffer, frameNum, offset) ;
	return buffer[addr] ; 	
}

__device__ void Gwrite( uchar *buffer, u32 addr, uchar value){
	/* Complete Gwrite function to write value to data buffer */
	u32 frameNum = addr/PAGESIZE ; 
	u32 offset =  addr%PAGESIZE ; 

	addr = paging(buffer, frameNum , offset) ; 
	buffer[addr] = value ; 
}

__device__ void snapshot( uchar *results, uchar* buffer, int offset, int input_size ){
	/* Complete snapshot function to load elements from data to result */
	for(int i = 0; i < input_size; i++ ){
		results[i] = Gread(buffer, i + offset) ; 
	}
}

__global__ void mykernel( int input_size ){
	//take shared memory as physical memory 
	__shared__ uchar data[PHYSICAL_MEM_SIZE];

	//get page table entries 
	int pt_entries = PHYSICAL_MEM_SIZE/PAGESIZE;
	
	//before first Gwrite or Gread 
	init_pageTable(pt_entries); 

	/* Gwrite / Gread starts */
	for(int i = 0; i < input_size; i++)
		Gwrite(data, i , input[i]); 
	
	for(int i = input_size - 1; i >= input_size - 32769; i-- )
		int value = Gread(data, i) ;

	snapshot( results, data, 0, input_size);

	printf("pagefault number is %d\n", PAGEFAULT_NUM) ;  
}

__device__ u32 getPageNum(u32 pt_entries){
	return (pt_entries & PAGENUMMASK) >>1;
}

__device__ u32 paging( uchar *buffer, u32 frameNum, u32 offset){
	u32 target ; 
	int pt_entries = PT_ENTRIES ;

	//Find if the target page exists
	for(int i = 0; i < pt_entries;i++){
		u32 pageNum = getPageNum(pt[i]); //Stores the logic page number of pt[i]

		if (pt[i] & 1==VALID){
			if (pageNum==frameNum){
				u32 tempTime=currentTime++;
				//update hit time
				pt[i]=(tempTime<<13)|(frameNum<<1)|VALID;
				return i*PAGESIZE+offset;
			}
		}
	}
	
	//Find if there is an empty entry 
	for(int i = 0; i < pt_entries;i++){
		//Bitwise checking the page table and find invalid entries and mark as pagefault
		if(pt[i] & 1==INVALID){
			PAGEFAULT_NUM++ ;
			//Update page table
			u32 tempTime = currentTime++ ;
			pt[i] = (tempTime << 13 ) | ( frameNum << 1 ) | 1 ; 
			return i * PAGESIZE + offset  ; 
		}
	}

	//Find a place for swapping in by the rule of LRU
	u32 leastTime=0xFFFFFFFF;
	for(int i = 0; i < pt_entries;i++){
		u32 mask = (u32)(-1);
		u32 currentTime  = (mask & pt[i]) >> 13 ;
		if(currentTime < leastTime){
			//The entry of the potential page to be swapped out
			target = i ;
			leastTime=currentTime;
		}
	}
	
	PAGEFAULT_NUM++ ;

	//The page number of the logical page to be swapped out
	u32 tarFrame = getPageNum(pt[target]);

	//The address of the target to be swapped out in the secondary memory
	u32 beginAddress = tarFrame * PAGESIZE; 
	for(int i = beginAddress, j = 0; j < PAGESIZE; i++, j++){
		//The address of the target to be swapped in in the physical memory
		u32 sharedAddress = target * PAGESIZE + j; 
		u32 curAddress = frameNum * PAGESIZE + j; 
		
		//Swap out
		storage[i] = buffer[sharedAddress];
		//Swap in		
		buffer[sharedAddress] = storage[curAddress];	
	}
	int tempTime = currentTime++ ; 
	pt[target] = ((tempTime) << 13 ) | ( frameNum << 1 ) | 1 ;
	return target * PAGESIZE + offset ;
}

__host__ void write_binaryFile(char *fileName, uchar *results, int bufferSize){
	FILE *fp;
	fp = fopen(fileName, "wb");
 
	fwrite(results,sizeof(uchar),bufferSize,fp);
	fclose(fp) ; 
}

__host__ int load_binaryFile(char *fileName, uchar *input, int bufferSize){
	FILE *fp;
	fp = fopen(fileName, "rb");

	if (!fp){
		printf("***Unable to open file %s***\n", fileName);
		exit(1);
	}

	fseek(fp, 0, SEEK_END);
	int fileLen = ftell(fp);
	
	fseek(fp, 0, SEEK_SET);

	//Read data from input file
	fread(input,sizeof(uchar),fileLen,fp); 

	if (fileLen > bufferSize){
		printf("****invalid testcase!!****\n");
		printf("****software warrning: the file: %s size****\n", fileName);
		printf("****is greater than buffer size****\n");
		exit(1);
	}

	fclose(fp);
	
	return fileLen;  
}

int main(){
	hipError_t cudaStatus;
	int input_size = load_binaryFile(DATAFILE, input, STORAGE_SIZE);

	mykernel<<<1, 1, 16384>>>(input_size);
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "mykernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return 0;
	}

	printf("input size: %d\n", input_size);

	hipDeviceSynchronize() ; 
	hipDeviceReset() ;

	write_binaryFile(OUTFILE, results, input_size);
	
	return 0 ; 
}
